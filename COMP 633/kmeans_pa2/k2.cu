//boilerplate includes from previous CUDA project

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <fcntl.h>
#include <stdlib.h>
#include <math.h>
#include <limits.h>

#include <string>
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include <utility>
#include <set>
#include <chrono>
#include <iomanip>

using namespace std;
using namespace std::chrono;

struct record {
  double x;
  double y;
  double z;
};

struct centroid {
    int member_count;
    record center;
    record member_sum;
};

void printRecord(record r){
    cout << fixed << "(" << r.x << ", " << r.y << ", " << r.z << ")" << endl;
}

void printCentroid(centroid c){
    cout << "Center: ";
    printRecord(c.center);

    cout << "Member Sum: ";
    printRecord(c.member_sum);

    cout << "Member count: " << c.member_count << endl;

}

void readfile(string path, vector<record> &records){
    
    ifstream infile;
    infile.open(path);
    double x, y, z;
    while (infile >> x >> y >> z){
        record curr;
        curr.x = x;
        curr.y = y;
        curr.z = z;
        records.push_back(curr);
    }
}

void initCentroid(centroid &c){
    c.member_count = 0;
    c.member_sum.x = 0;
    c.member_sum.y = 0;
    c.member_sum.z = 0;

    c.center.x = 0;
    c.center.y = 0;
    c.center.z = 0;
}

void centroidCenter(centroid &c){
    c.center.x = c.member_sum.x / (c.member_count * 1.0);
    c.center.y = c.member_sum.y / (c.member_count * 1.0);
    c.center.z = c.member_sum.z / (c.member_count * 1.0);
}

void addToMemberSum(centroid &c, record r){
    c.member_sum.x += r.x;
    c.member_sum.y += r.y;
    c.member_sum.z += r.z;
}

void removeFromMemberSum(centroid &c, record r){
    c.member_sum.x -= r.x;
    c.member_sum.y -= r.y;
    c.member_sum.z -= r.z;
}

double normalizerecord(record r){
    double total = 0.0;
    total += (r.x * r.x) + (r.y * r.y) + (r.z * r.z);
    return sqrt(total);
}

//distance between two location records
double vectordistance(record r1, record r2){
    double x = r1.x - r2.x;
    double y = r1.y - r2.y;
    double z = r1.z - r2.z;
    double dist = (x * x) + (y * y) + (z * z);
    return sqrt(dist);
}

void printAllRecords(vector<record> data){
    int i;
    for(i = 0; i < data.size(); i++){
        printRecord(data[i]);
    }
}

void init(int n, int k, vector<record> data, vector<int> &assignments,  vector<centroid> &centroids){
  
    int i;

    //default all centroids to 0 (so we don't get errs with adding members later)
    for(i = 0; i < k; i++){
        initCentroid(centroids[i]);
        //centroidCenter(centroids[i]);
    }

    //defaulting all feature vectors to cluster 0
    for(i = 0; i < n; i++){
        assignments[i] = 0;
        addToMemberSum(centroids[0], data[i]);
    }

    centroids[0].member_count = n;
    //centroidCenter(centroids[0]);

}

void altinit(int n, int k, vector<record> data, vector<int> &assignments,  vector<centroid> &centroids){
  
    int i;

    //default all centroids to 0 (so we don't get errs with adding members later)
    for(i = 0; i < k; i++){
        initCentroid(centroids[i]);
        //centroidCenter(centroids[i]);
    }

    int range = ceil(n / k);
    int rangecounter = 0;
    int rangecurr = 0;

    //defaulting all feature vectors to cluster 0
    for(i = 0; i < n; i++){

        if(rangecounter > range){
            rangecounter = 0;
            rangecurr++;
        }

        assignments[i] = rangecurr;
        addToMemberSum(centroids[rangecurr], data[i]);
        centroids[rangecurr].member_count++;

        rangecounter++;
    }

    //cout << "ON INIT" << endl;
    for(i = 0; i < k; i++){
        centroidCenter(centroids[i]);
        //printCentroid(centroids[i]);
        //printRecord(centroids[i].center);
    }

    //cout << "-------------------" << endl;

}

int getNearestCentroid(int k, record data, vector<centroid> centroids){

    int cmin = 0;
    int i;

    double dmin = INT_MAX;//((1 << 32) - 1); //shift to max int
    double d;

    for(i = 0; i < k; i++){
        d = vectordistance(data, centroids[i].center);
        if(d < dmin){
            dmin = d;
            cmin = i;
        }
    }

    return cmin;
}

int main(int argc, char **argv){

    //using our data set of ~8000 location points for our test
    //3-dimmensional features (x,y,z ECF coordinates)
    //all locational points are from same group of ~50 projectiles at different times
    vector<record> data;
    readfile("locations.big", data);

    //define everything
    int k = 225;
    int n = data.size();

    vector<int> assignments(n); //define assignments as their own list rather than appending to the struct
    vector<centroid> centroids(k);

    altinit(n, k, data, assignments, centroids);
    //printAllRecords(data);

    bool converged = false;
    while(!converged){

        cout << "Not converged, finding centroids..." << endl;

        converged = true;
        int cmin, i;
        for(i = 0; i < n; i++){
            cmin = getNearestCentroid(k, data[i], centroids);

            if(cmin != assignments[i]){

                converged = false;
                int prevCenIndex = assignments[i];

                //update membership count, sum
                removeFromMemberSum(centroids[prevCenIndex], data[i]);
                addToMemberSum(centroids[cmin], data[i]);
                centroids[prevCenIndex].member_count--;
                centroids[cmin].member_count++;

                assignments[i] = cmin;
            }

        }

        //update center of centroids
        for(i = 0; i < k; i++){
            centroidCenter(centroids[i]);
        }

    }

    int i;
    for(i = 0; i < n; i++){
        //cout << assignments[i] << endl;
    }

    return 0;
}